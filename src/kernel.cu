#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "gray_img_CUDA.cuh"

__global__ void gray_CUDA(unsigned char* orig, unsigned char* gray, int width, int height, int channels){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < width && y < height) {
        int grayOffset = y * width + x;
        int rgbOffset = grayOffset * channels;
        unsigned char r = orig[rgbOffset]; // red value for pixel
        unsigned char g = orig[rgbOffset + 1]; // green value for pixel
        unsigned char b = orig[rgbOffset + 2]; // blue value for pixel

        gray[rgbOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
        gray[rgbOffset + 1] = 0.21f*r + 0.71f*g + 0.07f*b;
        gray[rgbOffset + 2] = 0.21f*r + 0.71f*g + 0.07f*b;
    }
}

void grayParallel(unsigned char* orig, unsigned char* gray, int height, int width, int channels){
    unsigned char* gpu_data, *gray_data;

    //gpu memory allocate
    hipMalloc((void **) &gpu_data, height * width * channels * sizeof(unsigned char));
    hipMalloc((void **) &gray_data, height * width * channels * sizeof(unsigned char));

    //copy data from CPU to GPU
    hipMemcpy(gpu_data, orig, height * width * channels * sizeof(unsigned char), hipMemcpyHostToDevice);

    dim3 blockSize( (width-1)/16 + 1, (height-1)/16 + 1, 1);
    dim3 gridSize( 16, 16, 1);

    gray_CUDA<<<blockSize,gridSize>>>(gpu_data, gray_data, width, height, channels);

    //copy data from GPU to CPU
    hipMemcpy(gray, gray_data, height * width * channels * sizeof(unsigned char), hipMemcpyDeviceToHost);

    //free GPU memory
    hipFree(gpu_data);
    hipFree(gray_data);
}